#include "shapes/Sphere.cu"
#include "shapes/Plane.cu"
#include "shapes/Triangle.cu"

#include "integration/Scene.cu"
#include "integration/PathTracerIntegrator.cu"
#include "integration/photonDispersion.cu"

#include "camera/ProjectiveCamera.cu"
#include "sampling/Sampler.cu"
#include "light/PointLight.cu"
#include "materials/LambertianMaterial.cu"
#include "materials/LightMaterial.cu"
#include "materials/GlassMaterial.cu"
#include "materials/MirrorMaterial.cu"
#include "materials/SSMaterial.cu"
#include "materials/PlasticUnrealMaterial.cu"

#include "textures/Texture.cu"
#include "textures/SphericalTexture.cu"

#include "cuda_libraries/FileImporting.cu"

int main(int argc, char *argv[])
{
    std::string fileName = "image";

    // Get file name
    if (argc >= 2)
    {
        fileName = argv[1];
    }

    // Get gpu id
    if (argc == 3)
    {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);

        int gpuId = atoi(argv[2]);

        if (gpuId >= deviceCount)
        {
            std::cout << "Invalid GPU id" << std::endl;
            return 0;
        }

        hipSetDevice(gpuId);
    }

    // Create a sphere
    auto sphere1 = Sphere(Point3f(2.5, 1.25, 0), 
                        Vector3f(0, 0, 2), 
                        Point3f(2.5, 2.25, 0));

    auto sphere2 = Sphere(Point3f(2.5, 0, 0), 2);

    auto sphereL = Sphere(Point3f(0, -3.5, 0), 
                        Vector3f(0, 0, 4),
                        Point3f(0, -5.5, 0));

    auto worldSphere = Sphere(Point3f(0, 0, 0), 1);


    auto plane1 = Plane(Point3f(4, -4, -4), 
                        Vector3f(0, 8, 0), 
                        Vector3f(0, 0, 8), 
                        Vector3f(-4, 0, 0));

    auto plane2 = Plane(Point3f(4, 4, -4), 
                        Vector3f(-8, 0, 0), 
                        Vector3f(0, 0, 8), 
                        Vector3f(0, -4, 0));

    auto plane3 = Plane(Point3f(-4, -4, -4), 
                        Vector3f(0, 8, 0), 
                        Vector3f(0, 0, 8), 
                        Vector3f(4, 0, 0));

    auto plane4 = Plane(Point3f(4, -4, -4), 
                        Vector3f(-8, 0, 0), 
                        Vector3f(0, 0, 8), 
                        Vector3f(0, 4, 0));

    auto plane5 = Plane(Point3f(-4, -4, -4), 
                        Vector3f(8, 0, 0), 
                        Vector3f(0, 8, 0), 
                        Vector3f(0, 0, 4));

    auto plane6 = Plane(Point3f(-4, -4, 4), 
                        Vector3f(8, 0, 0), 
                        Vector3f(0, 8, 0), 
                        Vector3f(0, 0, -1));

    auto planeFront = Plane(Point3f(1, -1, -1), 
                        Vector3f(0, 8, 0), 
                        Vector3f(0, 0, 8), 
                        Vector3f(-1, 0, 0));

    LambertianMaterial blueMaterial(Spectrum(0, 0, 1.0f));
    LambertianMaterial greenMaterial(Spectrum(0.0f, 1.0f, 0.0f));
    LambertianMaterial redMaterial(Spectrum(1.0f, 0.0f, 0.0f));
    LambertianMaterial whiteMaterial(Spectrum(1.0f, 1.0f, 1.0f));
    LambertianMaterial greyMaterial(Spectrum(0.7f, 0.7f, 0.7f));
    SSMaterial marbleMaterial(Spectrum(0), Spectrum(0.83, 0.79, 0.75), Spectrum(0), 1.3, 0, nullptr, false, 
            Spectrum(2.19, 2.62, 3), Spectrum(0.0021, 0.00041, 0.0071), 0);
    LambertianMaterial marbleBRDF(Spectrum(0.83, 0.79, 0.75));

    LightMaterial lightMaterial(Spectrum(1, 1, 1), 4);
    DielectricMaterial glassMaterial(Spectrum(185.0/255, 232.0/255, 234.0/255), 1.31);
    MirrorMaterial mirror(Spectrum(1, 1, 1));
    PlasticUnrealMaterial plastic(Spectrum(1, 1, 1), 1);

    std::shared_ptr<std::vector<TriangleMesh>> triangleMeshes = std::make_shared<std::vector<TriangleMesh>>();
    std::vector<std::vector<Triangle>> triangles;
    TriangleMesh mesh1;
    std::vector<Triangle> mesh1Triangles;

    loadObj("./models/basics/sphere.obj", triangleMeshes, triangles);
    //loadObj("/export/d03/scratch/a816678/pathtracing_renderer/models/basics/cube.obj", triangleMeshes, triangles);


    //Texture *texture = new Texture("/home/hsunekichi/Escritorio/pathtracing_renderer/bricks.bmp");
    Texture *texture = new Texture("./bricks.bmp");
    
    //Texture *sph_texture = new SphericalTexture("/home/hsunekichi/Escritorio/pathtracing_renderer/marsSPH.bmp");
    //Texture *sph_texture = new SphericalTexture("/home/hsunekichi/Escritorio/pathtracing_renderer/marsSPH.bmp");

    LambertianMaterial bricksMaterial(Spectrum(1), Spectrum(1), texture);
    //LambertianMaterial marsMaterial(Spectrum(1), Spectrum(1), sph_texture);

    sphere1.setMaterial(&whiteMaterial);
    sphere2.setMaterial(&plastic);
    worldSphere.setMaterial(&whiteMaterial);


    plane1.setMaterial(&whiteMaterial);
    plane2.setMaterial(&greenMaterial);
    plane3.setMaterial(&whiteMaterial);
    plane4.setMaterial(&redMaterial);
    plane5.setMaterial(&whiteMaterial);
    plane6.setMaterial(&whiteMaterial);
    planeFront.setMaterial(&glassMaterial);

    //sphereL.setMaterial(&lightMaterial);

    // Create device array with the shape*
    std::vector<Shape*> shapes;
    //shapes.push_back(&sphere1);
    //shapes.push_back(&sphere2);
    //shapes.push_back(&plane2);
    //shapes.push_back(&worldSphere);

    shapes.push_back(&plane1);
    shapes.push_back(&plane2);
    //shapes.push_back(&plane3);
    shapes.push_back(&plane4);
    shapes.push_back(&plane5);
    shapes.push_back(&plane6);
    //shapes.push_back(&planeFront);

    std::vector<Light*> lights;
    Camera *camera;

    //loadObejota("/home/hsunekichi/Escritorio/pathtracing_renderer/box.obejota", lights, shapes, camera);
    loadObejota("./box.obejota", lights, shapes, camera);

    Scene scene;
    scene.build(shapes, lights, triangles, triangleMeshes);
   
    //PathTracerIntegrator integrator(&camera, &sampler);
    PathTracerIntegrator integrator(camera);

    integrator.render(&scene);
    Film *film = camera->getFilm();

    //film.filterIndirectLight();
    //film.loadContributionsFromFile("test");
    film->storeContributionsToFile("box");
    
    if (RESCALE_IMAGE == true)
        film->rescale(3840, 2160);
        

    film->writeToBMP("box", ToneMappingType::gammaClamp);
}