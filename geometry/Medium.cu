#pragma once
#include <hip/hip_runtime.h>


class Medium
{
    public:

    Medium** toGPU() const
    {
        // Allocate memory on gpu for pointer
        Medium** gpuPtr;
        hipMalloc(&gpuPtr, sizeof(Medium*));

        return gpuPtr;
    }
};